
#include "cudnn_helpers.cuh"
#ifdef HAVE_CUDNN
#include <hipDNN.h>
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>

#define CUDNN_CHECK(x) do { \
  hipdnnStatus_t s = (x); \
  if(s != HIPDNN_STATUS_SUCCESS){ \
    fprintf(stderr, "cuDNN error %s at %s:%d\n", hipdnnGetErrorString(s), __FILE__, __LINE__); \
    asm("trap;"); \
  } \
} while(0)

void cudnn_conv_relu_forward_nchw(
  const float* x, const float* w, const float* b, float* y,
  int N,int C_in,int H,int W,
  int C_out,int K_h,int K_w,
  int stride_h,int stride_w,
  int pad_h,int pad_w)
{
  hipdnnHandle_t h; CUDNN_CHECK(hipdnnCreate(&h));

  hipdnnTensorDescriptor_t xDesc, yDesc, bDesc;
  hipdnnFilterDescriptor_t wDesc;
  hipdnnConvolutionDescriptor_t convDesc;
  hipdnnActivationDescriptor_t actDesc;

  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&xDesc));
  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&yDesc));
  CUDNN_CHECK(hipdnnCreateTensorDescriptor(&bDesc));
  CUDNN_CHECK(hipdnnCreateFilterDescriptor(&wDesc));
  CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));
  CUDNN_CHECK(hipdnnCreateActivationDescriptor(&actDesc));

  // NCHW float
  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C_in, H, W));
  int H_out = (H + 2*pad_h - K_h)/stride_h + 1;
  int W_out = (W + 2*pad_w - K_w)/stride_w + 1;
  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C_out, H_out, W_out));
  CUDNN_CHECK(hipdnnSetFilter4dDescriptor(wDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, C_out, C_in, K_h, K_w));
  CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(convDesc, pad_h, pad_w, stride_h, stride_w, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

  // Choose algo
  hipdnnConvolutionFwdAlgo_t algo;
  CUDNN_CHECK(hipdnnGetConvolutionForwardAlgorithm(h, xDesc, wDesc, convDesc, yDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

  // Workspace
  size_t ws_bytes=0;
  CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(h, xDesc, wDesc, convDesc, yDesc, algo, &ws_bytes));
  void* ws=nullptr; if(ws_bytes) hipMalloc(&ws, ws_bytes);

  // Bias
  CUDNN_CHECK(hipdnnSetTensor4dDescriptor(bDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, C_out, 1, 1));

  // Activation (ReLU)
  CUDNN_CHECK(hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

  // conv
  const float alpha=1.f, beta=0.f;
  CUDNN_CHECK(hipdnnConvolutionForward(h, &alpha, xDesc, x, wDesc, w, convDesc, algo, ws, ws_bytes, &beta, yDesc, y));

  // bias add
  CUDNN_CHECK(hipdnnAddTensor(h, &alpha, bDesc, b, &alpha, yDesc, y));

  // relu in-place
  CUDNN_CHECK(hipdnnActivationForward(h, actDesc, &alpha, yDesc, y, &beta, yDesc, y));

  if(ws) hipFree(ws);
  hipdnnDestroyActivationDescriptor(actDesc);
  hipdnnDestroyConvolutionDescriptor(convDesc);
  hipdnnDestroyFilterDescriptor(wDesc);
  hipdnnDestroyTensorDescriptor(bDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroy(h);
}
#endif
